
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

void readPnm(char * fileName, 
		int &numChannels, int &width, int &height, uint8_t * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	if (strcmp(type, "P2") == 0)
		numChannels = 1;
	else if (strcmp(type, "P3") == 0)
		numChannels = 3;
	else // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);

	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uint8_t *)malloc(width * height * numChannels);
	for (int i = 0; i < width * height * numChannels; i++)
		fscanf(f, "%hhu", &pixels[i]);

	fclose(f);
}

void writePnm(uint8_t * pixels, int numChannels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	if (numChannels == 1)
		fprintf(f, "P2\n");
	else if (numChannels == 3)
		fprintf(f, "P3\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height * numChannels; i++)
		fprintf(f, "%hhu\n", pixels[i]);

	fclose(f);
}

__global__ void convertRgb2GrayKernel(uint8_t * inPixels, int width, int height, 
		uint8_t * outPixels)
{
	// TODO
	int iy = blockDim.y * blockIdx.y + threadIdx.y;
	int ix = blockDim.x * blockIdx.x + threadIdx.x;
	if (ix < width && iy <height)
	{
		int i = iy* width +ix;
		outPixels[i] = inPixels[3*i]*0.299 + inPixels[3*i +1]*0.587 + inPixels[3*i +2]*0.114;
	}
}

void convertRgb2Gray(uint8_t * inPixels, int width, int height,
		uint8_t * outPixels, 
		bool useDevice=false, dim3 blockSize=dim3(1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		for (int r = 0; r < height; r++)
		{
			for (int c = 0; c < width; c++)
			{
				int i = r * width + c;
				uint8_t r = inPixels[i * 3];
				uint8_t g = inPixels[i * 3 + 1];
				uint8_t b = inPixels[i * 3 + 2];
				outPixels[i] = 0.299f * r + 0.587f * g + 0.114f * b;
			}
		}
	}
	else // use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO: Allocate device memories
		uint8_t *d_inpixels, *d_outpixels;
		CHECK(hipMalloc(&d_inpixels, width * height * 3 * sizeof(uint8_t)));
		CHECK(hipMalloc(&d_outpixels, width * height * sizeof(uint8_t)));


		// TODO: Copy data to device memories
		
		CHECK(hipMemcpy(d_inpixels,inPixels,width * height*3 * sizeof(uint8_t),hipMemcpyHostToDevice));

		// TODO: Set grid size and call kernel (remember to check kernel error)
		dim3 gridSize((width-1)/blockSize.x + 1,(height-1)/blockSize.y + 1);

		convertRgb2GrayKernel<<<gridSize,blockSize>>>(d_inpixels,width,height,d_outpixels);
		hipDeviceSynchronize();
		CHECK(hipGetLastError());

		
		// TODO: Copy result from device memories
		CHECK(hipMemcpy(outPixels,d_outpixels, width*height*sizeof(uint8_t),hipMemcpyDeviceToHost));

		

		// TODO: Free device memories
		hipFree(d_outpixels);
		hipFree(d_inpixels);

		// for (int i = 0; i < width*height*3; i++)
		// 	printf("%d\t",d_inpixels[i]);

	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n\n", 
			useDevice == true? "use device" : "use host", time);
}

float computeError(uint8_t * a1, uint8_t * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
		err += abs((int)a1[i] - (int)a2[i]);
	err /= n;
	
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
	char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
	strcpy(result, s1);
	strcat(result, s2);
	return result;
}

int main(int argc, char ** argv)
{	
	if (argc != 3 && argc != 5)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input RGB image file
	int numChannels, width, height;
	uint8_t * inPixels;
	readPnm(argv[1], numChannels, width, height, inPixels);
	if (numChannels != 3)
		return EXIT_FAILURE; // Input image must be RGB
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Convert RGB to grayscale not using device
	uint8_t * correctOutPixels= (uint8_t *)malloc(width * height);
	convertRgb2Gray(inPixels, width, height, correctOutPixels);

	// Convert RGB to grayscale using device
	uint8_t * outPixels= (uint8_t *)malloc(width * height);
	dim3 blockSize(32, 32); // Default
	if (argc == 5)
	{
		blockSize.x = atoi(argv[3]);
		blockSize.y = atoi(argv[4]);
	} 
	convertRgb2Gray(inPixels, width, height, outPixels, true, blockSize); 

	// Compute mean absolute error between host result and device result
	float err = computeError(outPixels, correctOutPixels, width * height);
	printf("Error between device result and host result: %f\n", err);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(correctOutPixels, 1, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(outPixels, 1, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(outPixels);
}
