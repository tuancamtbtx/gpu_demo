
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, 
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
		{
			fclose(f);
			printf("Cannot read %s\n", fileName); 
			exit(EXIT_FAILURE); 
		}

		fscanf(f, "%i", &width);
		fscanf(f, "%i", &height);
		
		int max_val;
		fscanf(f, "%i", &max_val);
		if (max_val > 255) // In this exercise, we assume 1 byte per value
		{
			fclose(f);
			printf("Cannot read %s\n", fileName); 
			exit(EXIT_FAILURE); 
		}

		pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
		for (int i = 0; i < width * height; i++)
			fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

		fclose(f);
	}

	void writePnm(uchar3 * pixels, int width, int height, 
			char * fileName)
	{
		FILE * f = fopen(fileName, "w");
		if (f == NULL)
		{
			printf("Cannot write %s\n", fileName);
			exit(EXIT_FAILURE);
		}	

		fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

		for (int i = 0; i < width * height; i++)
			fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
		
		fclose(f);
	}

	__global__ void blurImgKernel(uchar3 * inPixels, int width, int height, 
			float * filter, int filterWidth, 
			uchar3 * outPixels)
	{
		// TODO
		int iy = blockDim.y*blockIdx.y + threadIdx.y;
		int ix = blockDim.x*blockIdx.x + threadIdx.x;
		int pos = iy* width + ix;

		if (ix < width && iy < height);
		{
			float x = 0;
			float y = 0;
			float z = 0;				
			for (int h=0 ; h<filterWidth ; h++) {
				for (int w=0 ; w<filterWidth ; w++) {
					
					int imagex = ix - filterWidth/2 + w;
					int imagey = iy - filterWidth/2 + h;
					if (h*filterWidth + w < 0 || h*filterWidth + w >= filterWidth*filterWidth)
						printf("%d-",h*filterWidth + w);
					if ( 0 <= imagex && imagex < width && 0 <= imagey && imagey < height){
												
						x += filter[h*filterWidth + w]*inPixels[imagey*width + imagex].x;
						y += filter[h*filterWidth + w]*inPixels[imagey*width + imagex].y;	
						z += filter[h*filterWidth + w]* inPixels[imagey*width + imagex].z;
					}
				}
			}
			
			outPixels[pos].x = x;
			outPixels[pos].y = y;
			outPixels[pos].z = z;
		}
}

void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
		uchar3 * outPixels,
		bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		// TODO 
		for (int i=0 ; i<height ; i++) {
			for (int j=0 ; j<width ; j++) {
				float x = 0;
				float y = 0;
				float z = 0;
				for (int h=0 ; h<filterWidth ; h++) {
					for (int w=0 ; w<filterWidth ; w++) {
						if (i -filterWidth/2 + h >= 0 && i -filterWidth/2 + h < height && j -filterWidth/2 + w >= 0 && j -filterWidth/2 + w <width)
						{
							x += filter[h*filterWidth + w]*inPixels[(i-filterWidth/2 + h)*width + (j - filterWidth/2 + w)].x;
							y += filter[h*filterWidth + w]*inPixels[(i-filterWidth/2 + h)*width + (j - filterWidth/2 + w)].y;
							z += filter[h*filterWidth + w]*inPixels[(i-filterWidth/2 + h)*width + (j - filterWidth/2 + w)].z;
						}
						
					}
				}
				outPixels[i*width+j].x = x;
				outPixels[i*width+j].y = y;
				outPixels[i*width+j].z = z;
			}
		}

	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO
		uchar3 * d_inPixels, * d_outPixels;
		float *d_filter;
		CHECK(hipMalloc(&d_inPixels, width*height*sizeof(uchar3)));
		CHECK(hipMalloc(&d_outPixels, width*height*sizeof(uchar3)));
		CHECK(hipMalloc(&d_filter, filterWidth*filterWidth*sizeof(float)));
		
		CHECK(hipMemcpy(d_inPixels,inPixels,width*height*sizeof(uchar3),hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_filter,filter,filterWidth*filterWidth*sizeof(float),hipMemcpyHostToDevice));

		dim3 gridSize((width-1)/blockSize.x +1,(height-1)/blockSize.y +1 );

		blurImgKernel<<<gridSize,blockSize>>>(d_inPixels,width,height,d_filter,filterWidth,d_outPixels);
		hipDeviceSynchronize();
		CHECK(hipGetLastError());

		CHECK(hipMemcpy(outPixels,d_outPixels,width*height*sizeof(uchar3),hipMemcpyDeviceToHost));

		hipFree(d_outPixels);
		hipFree(d_inPixels);
		hipFree(d_filter);

	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n", 
    		useDevice == true? "use device" : "use host", time);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		// printf()
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main(int argc, char ** argv)
{
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Read correct output image file
	int correctWidth, correctHeight;
	uchar3 * correctOutPixels;
	readPnm(argv[3], correctWidth, correctHeight, correctOutPixels);
	if (correctWidth != width || correctHeight != height)
	{
		printf("The shape of the correct output image is invalid\n");
		return EXIT_FAILURE;
	}

	// Set up a simple filter with blurring effect 
	int filterWidth = 9;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth*filterWidth);
		}
	}

	// Blur input image using host
	uchar3 * hostOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, hostOutPixels);
	
	// Compute mean absolute error between host result and correct result
	float hostErr = computeError(hostOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", hostErr);

	// Blur input image using device
	uchar3 * deviceOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	dim3 blockSize(32, 32); // Default
	if (argc == 6)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}  
	blurImg(inPixels, width, height, filter, filterWidth, deviceOutPixels, true, blockSize);

	// Compute mean absolute error between device result and correct result
	float deviceErr = computeError(deviceOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", deviceErr);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(correctOutPixels);
	free(hostOutPixels);
	free(deviceOutPixels);
	free(filter);
}
